#include <iostream>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <math.h>
#include <hiprand/hiprand.h>

using namespace std;
#define CUDA_CALL(x) do { \
      hipError_t _m_cudaStat = x; \
      if((_m_cudaStat) != hipSuccess) { \
      fprintf(stderr, "Error %s at line %d in file %s", \
      hipGetErrorString(_m_cudaStat), __LINE__, __FILE__); \
      exit(1);}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
      printf("CURAND rrror at %s:%d\n",__FILE__,__LINE__);            \
      exit(1);}} while(0)

__global__ void cnat_compress_cuda_kernel(
    float* __restrict__ input,
    uint8_t* __restrict__ output,
    int len) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < len) {
    if (input[index] == 0)
      output[index] = 0;
    else {
      int rand_bits = output[index];
      rand_bits <<= 15;
      int exp;
      int prob = reinterpret_cast<int &>(input[index]) & 0b00000000011111111000000000000000;
      frexpf(input[index], &exp);
      if (rand_bits >= prob) exp -= 1;
      exp += 127;
      uint8_t encode;
      if (exp<=17) encode = 0;
      else if (exp<=143) encode = uint8_t(exp-17);
      else encode = 127;
      if (input[index] < 0) encode += 128;
      output[index] = encode;
    }
  }
}

/*
  generate random bits using the output buffer (8 bits for each element)
  and compare with the 8 most significant bits of mantissa
*/
void cnat_compress(float* input, uint8_t* output, int count, hipStream_t stream, hiprandGenerator_t* gen) {
    const int threads = 1024;
    auto blocks = count/threads;
    if (count%threads || !blocks) blocks++;
    CURAND_CALL(hiprandSetStream(*gen, stream));
    CURAND_CALL(hiprandGenerate(*gen, (uint32_t*)output, count/4)); // (0, 1]
    cnat_compress_cuda_kernel<<<blocks, threads, 0, stream>>>(
            input,
            output,
            count);
}

__global__ void cnat_decompress_cuda_kernel(
    uint8_t* __restrict__ input,
    float* __restrict__ output,
    int len) {
  const int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < len) {
      uint32_t decode;
      if (input[index]<=127) decode = input[index]+17;
      else decode = input[index]+145;
      if (!input[index] % 128) decode -= 17;
      uint32_t sign_and_exp = decode << 23;
      output[index] = reinterpret_cast<float &>(sign_and_exp);
  }
}

void cnat_decompress(uint8_t* input, float* output, int count, hipStream_t stream) {
  const int threads = 1024;
  auto blocks = count/threads;
  if (count%threads || !blocks) blocks++;
  cnat_decompress_cuda_kernel<<<blocks, threads, 0, stream>>>(
    input,
    output,
    count);
}

int main(){
    hiprandGenerator_t gen;
    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));
    CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL)));
    float host[] = {3., 3., 3., 3., 3., 3., 3., 3.};
    float* ptr;
    uint8_t* output;
    uint8_t host_output[8];
    for(int i=0; i<8; ++i) cout<<host[i]<<", ";
    cout<<endl;
    CUDA_CALL(hipMalloc((void **)&ptr, 8*sizeof(float)));
    CUDA_CALL(hipMalloc((void **)&output, 8));
    hipMemcpy(ptr, host, 8*sizeof(float), hipMemcpyHostToDevice);
    cnat_compress(ptr, output, 8, stream, &gen);
    cnat_decompress(output, ptr, 8, stream);
    hipStreamSynchronize(stream);
    hipMemcpy(host_output, output, 8, hipMemcpyDeviceToHost);
    hipMemcpy(host, ptr, 32, hipMemcpyDeviceToHost);
    hipStreamSynchronize(stream);
    for(int i=0; i<8; ++i) cout<<int(host_output[i])<<", ";
    cout<<endl;
    for(int i=0; i<8; ++i) cout<<(host[i])<<", ";
    cout<<endl;
    CURAND_CALL(hiprandDestroyGenerator(gen));
    return 0;
}
